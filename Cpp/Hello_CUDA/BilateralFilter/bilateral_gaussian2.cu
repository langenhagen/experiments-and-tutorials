#include "hip/hip_runtime.h"
/******************************************************************************
/* @file Impl of bilateral_gaussian.cuh
/* 
/* TODO rename the file maybe
/*
/* @author langenhagen
/* @version 141210
/******************************************************************************/

///////////////////////////////////////////////////////////////////////////////
// INCLUDES project headers

#include "bilateral_gaussian2.cuh"

///////////////////////////////////////////////////////////////////////////////
//INCLUDES C/C++ standard library (and other external libraries)

#include <math.h>  // expf
#include <cstdlib> // malloc


///////////////////////////////////////////////////////////////////////////////
// KERNEL FUNCTIONS 


/**
 */
template<int RADIUS, int CHANNELS> __global__
void bilateral_gaussian2_kernel( float* m,
                                 float* o,
                                 unsigned int rows,
                                 unsigned int cols,
                                 float* spacial_convolution_kernel,
                                 const float inv_two_color_variance) {                                    

#if defined(KERNEL_WIDTH)
#error Macro KERNEL_WIDTH is already defined.
#endif

#define KERNEL_WIDTH (RADIUS+RADIUS+1)


    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // collaboratively create the kernel
    __shared__ float kernel_s[KERNEL_WIDTH][KERNEL_WIDTH];
    kernel_s[ty][tx] = spacial_convolution_kernel[ ty*KERNEL_WIDTH + tx];
    __syncthreads();


    cols *= CHANNELS;
    const int r = (blockIdx.y*KERNEL_WIDTH + ty);
	const int c = (blockIdx.x*KERNEL_WIDTH + tx)*CHANNELS + threadIdx.z;
    
    if( r >= rows || c >= cols)
		return;

    const int height = rows-1;
    const int width  = cols-CHANNELS;
    
    const float old_v = m[r*cols+c];
    float v(0.0f);
    float normalization(0);

    for (int i=-RADIUS; i<=RADIUS; ++i)   // rows
	for (int j=-RADIUS; j<=RADIUS; ++j) { // cols
			
        // clamp filter to image borders
        const float m_v = m[min(max(r+i, 0), height)*cols + min(max(c+j*CHANNELS, tz), width+tz)];
        
        float weight = /*spacial part*/ kernel_s[i+RADIUS][j+RADIUS] * 
                       /* color part */ __expf( -((m_v-old_v)*(m_v-old_v)) * inv_two_color_variance);

		v += m_v * weight;
        normalization += weight;
	}

    o[r*cols+c] = v/normalization;

#undef KERNEL_WIDTH
}


///////////////////////////////////////////////////////////////////////////////
// HELPER FUNCTIONS 

/**
 */
__host__
float* prepare_spacial_kernel2( int kernel_radius, float sigma) {
    const int width(kernel_radius+kernel_radius+1);
    float* ret = (float*)malloc( width*width*sizeof(float));
    float sum(0);
    
    // calculate weights
    float reciprocal_two_sigma_squared = 1/(2.0f*sigma*sigma);
    for( int r=-kernel_radius; r<=kernel_radius; ++r)
    for( int c=-kernel_radius; c<=kernel_radius; ++c) {
    
        float weight = expf( -(c*c+r*r) * reciprocal_two_sigma_squared);
        int idx = (r+kernel_radius)*width + c+kernel_radius;
        ret[idx] = weight;
        sum += weight;
    }
    
    // normalize
    sum = 1.0f/sum;
    for( int i=0; i<width*width; ++i)
        ret[i] *= sum;
    

    return ret;
}


///////////////////////////////////////////////////////////////////////////////
// TEST FUNCTIONS


/**
 */
__host__
void test_bilateral_gaussian2_9x9x3( float* m,
                                     float* o,
                                     const unsigned int rows,
                                     const unsigned int cols,
                                     const float spacial_sigma,
                                     const float color_sigma) {
#define RADIUS_ 4
#define WIDTH_ (RADIUS_ + RADIUS_ + 1)
#define CHANNELS_ 3

    const int kernel_radius = 4;
    const int kernel_width = kernel_radius+kernel_radius+1;

    float *d_m, *d_o, *kernel, *d_kernel;
    int m_size = rows*cols*CHANNELS_*sizeof(float);
    int kernel_size = kernel_width*kernel_width*sizeof(float);

    kernel = prepare_spacial_kernel2( kernel_radius, spacial_sigma);

    hipMalloc( (void**)&d_m, m_size);
    hipMemcpy( d_m, m, m_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_kernel, kernel_size);
    hipMemcpy( d_kernel, kernel, kernel_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_o, m_size);


    const int block_width = kernel_width;
    const dim3 dimBlock( block_width, block_width, 3);
    const dim3 dimGrid( (cols-1)/block_width+1, (rows-1)/block_width+1);
    

    bilateral_gaussian2_kernel<4,3><<<dimGrid, dimBlock>>>(
        d_m,
        d_o,
        rows,
        cols,
        d_kernel,
        1.0f/( 2*color_sigma*color_sigma));


    hipMemcpy( o, d_o, m_size, hipMemcpyDeviceToHost);

    hipFree( d_m);
    hipFree( d_o);
    hipFree( d_kernel);

#undef RADIUS_
#undef WIDTH_
#undef CHANNELS_
}




/**
 */
__host__
void test_bilateral_gaussian2_17x17x3( float* m,
                                       float* o,
                                       const unsigned int rows,
                                       const unsigned int cols,
                                       const unsigned int n_channels,
                                       const float spacial_sigma,
                                       const float color_sigma) {

#define RADIUS_ 4
#define WIDTH_ (RADIUS_ + RADIUS_ + 1)
#define CHANNELS_ 3

    const int kernel_radius = 8;
    const int kernel_width = kernel_radius+kernel_radius+1;

    float *d_m, *d_o, *kernel, *d_kernel;
    int m_size = rows*cols*n_channels*sizeof(float);
    int kernel_size = kernel_width*kernel_width*sizeof(float);

    kernel = prepare_spacial_kernel2( kernel_radius, spacial_sigma);

    hipMalloc( (void**)&d_m, m_size);
    hipMemcpy( d_m, m, m_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_kernel, kernel_size);
    hipMemcpy( d_kernel, kernel, kernel_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_o, m_size);


    const int block_width = kernel_width;
    const dim3 dimBlock( block_width, block_width, 3);
    const dim3 dimGrid( (cols-1)/block_width+1, (rows-1)/block_width+1);
    

    bilateral_gaussian2_kernel<8,3><<<dimGrid, dimBlock>>>(
        d_m,
        d_o,
        rows,
        cols,
        d_kernel,
        1.0f/( 2*color_sigma*color_sigma));


    hipMemcpy( o, d_o, m_size, hipMemcpyDeviceToHost);

    hipFree( d_m);
    hipFree( d_o);
    hipFree( d_kernel);

#undef RADIUS_
#undef WIDTH_
#undef CHANNELS_
}