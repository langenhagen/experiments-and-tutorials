#include "hip/hip_runtime.h"
/******************************************************************************
/* @file Impl of bilateral_gaussian.cuh
/* 
/* @author langenhagen
/* @version 141209
/******************************************************************************/

///////////////////////////////////////////////////////////////////////////////
// INCLUDES project headers

#include "bilateral_gaussian.cuh"

///////////////////////////////////////////////////////////////////////////////
//INCLUDES C/C++ standard library (and other external libraries)

#include <math.h>  // expf
#include <cstdlib> // malloc


///////////////////////////////////////////////////////////////////////////////
// KERNEL FUNCTIONS 


/**
 */
template<int RADIUS, int CHANNELS> __global__
void bilateral_gaussian_kernel( float* m,
                                float* o,
                                unsigned int rows,
                                unsigned int cols,
                                float* spacial_convolution_kernel,
                                const float inv_two_color_variance) {                                    

#if defined(KERNEL_WIDTH)
#error Macro KERNEL_WIDTH is already defined.
#endif

#define KERNEL_WIDTH (RADIUS+RADIUS+1)

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;

    // collaboratively create the kernel
    __shared__ float kernel_s[KERNEL_WIDTH][KERNEL_WIDTH];
    kernel_s[ty][tx] = spacial_convolution_kernel[ ty*KERNEL_WIDTH + tx];
    __syncthreads();


    cols *= CHANNELS;
    const int r = (blockIdx.y*KERNEL_WIDTH + ty);
	const int c = (blockIdx.x*KERNEL_WIDTH + tx)*CHANNELS + tz;
    
    if( r >= rows || c >= cols)
		return;

    const int height = rows-1;
    const int width  = cols-CHANNELS;
    
    const float old_v = m[r*cols+c];
    float v(0.0f);
    float normalization(0);

#pragma unroll
    for (int i=-RADIUS; i<=RADIUS; ++i)   // rows
#pragma unroll
	for (int j=-RADIUS; j<=RADIUS; ++j) { // cols
			
        // clamp filter to image borders
        const float m_v = m[min(max(r+i, 0), height)*cols + min(max(c+j*CHANNELS, tz), width+tz)];
        
        float weight = /*spacial part*/ kernel_s[i+RADIUS][j+RADIUS] * 
                       /* color part */ __expf( -((m_v-old_v)*(m_v-old_v)) * inv_two_color_variance);

		v += m_v * weight;
        normalization += weight;
	}

    o[r*cols+c] = v/normalization;

#undef KERNEL_WIDTH
}


///////////////////////////////////////////////////////////////////////////////
// HELPER FUNCTIONS 

/**
 */
__host__
float* prepare_spacial_kernel( int kernel_radius, float sigma) {
    const int width(kernel_radius+kernel_radius+1);
    float* ret = (float*)malloc( width*width*sizeof(float));
    float sum(0);
    
    // calculate weights
    float reciprocal_two_sigma_squared = 1/(2.0f*sigma*sigma);
    for( int r=-kernel_radius; r<=kernel_radius; ++r)
    for( int c=-kernel_radius; c<=kernel_radius; ++c) {
    
        float weight = expf( -(c*c+r*r) * reciprocal_two_sigma_squared);
        int idx = (r+kernel_radius)*width + c+kernel_radius;
        ret[idx] = weight;
        sum += weight;
    }
    
    // normalize
    sum = 1.0f/sum;
    for( int i=0; i<width*width; ++i) {
        ret[i] *= sum;
    }

    return ret;
}


///////////////////////////////////////////////////////////////////////////////
// TEST FUNCTIONS


/**
 */
__host__
void test_bilateral_gaussian_9x9x3( float* m,
                                    float* o,
                                    const unsigned int rows,
                                    const unsigned int cols,
                                    const float spacial_sigma,
                                    const float color_sigma) {

#define RADIUS_ 4
#define WIDTH_ (RADIUS_ + RADIUS_ + 1)
#define CHANNELS_ 3


    float *d_m, *d_o, *kernel, *d_kernel;
    int m_size = rows*cols*CHANNELS_*sizeof(float);
    int kernel_size = WIDTH_*WIDTH_*sizeof(float);

    kernel = prepare_spacial_kernel( RADIUS_, spacial_sigma);

    hipMalloc( (void**)&d_m, m_size);
    hipMemcpy( d_m, m, m_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_kernel, kernel_size);
    hipMemcpy( d_kernel, kernel, kernel_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_o, m_size);


    const dim3 dimBlock( WIDTH_, WIDTH_, CHANNELS_);
    const dim3 dimGrid( (cols-1)/WIDTH_+1, (rows-1)/WIDTH_+1);
    

    bilateral_gaussian_kernel<RADIUS_, CHANNELS_><<<dimGrid, dimBlock>>>(
        d_m,
        d_o,
        rows,
        cols,
        d_kernel,
        1.0f/( 2*color_sigma*color_sigma));


    hipMemcpy( o, d_o, m_size, hipMemcpyDeviceToHost);

    hipFree( d_m);
    hipFree( d_o);
    hipFree( d_kernel);

#undef RADIUS_
#undef WIDTH_
#undef CHANNELS_
}




/**
 */
__host__
void test_bilateral_gaussian_17x17x3( float* m,
                                      float* o,
                                      const unsigned int rows,
                                      const unsigned int cols,
                                      const float spacial_sigma,
                                      const float color_sigma) {

#define RADIUS_ 8
#define WIDTH_ (RADIUS_ + RADIUS_ + 1)
#define CHANNELS_ 3


    float *d_m, *d_o, *kernel, *d_kernel;
    int m_size = rows*cols*CHANNELS_*sizeof(float);
    int kernel_size = WIDTH_*WIDTH_*sizeof(float);

    kernel = prepare_spacial_kernel( RADIUS_, spacial_sigma);

    hipMalloc( (void**)&d_m, m_size);
    hipMemcpy( d_m, m, m_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_kernel, kernel_size);
    hipMemcpy( d_kernel, kernel, kernel_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_o, m_size);


    const dim3 dimBlock( WIDTH_, WIDTH_, CHANNELS_);
    const dim3 dimGrid( (cols-1)/WIDTH_+1, (rows-1)/WIDTH_+1);
    

    bilateral_gaussian_kernel<RADIUS_, CHANNELS_><<<dimGrid, dimBlock>>>(
        d_m,
        d_o,
        rows,
        cols,
        d_kernel,
        1.0f/( 2*color_sigma*color_sigma));


    hipMemcpy( o, d_o, m_size, hipMemcpyDeviceToHost);

    hipFree( d_m);
    hipFree( d_o);
    hipFree( d_kernel);

#undef RADIUS_
#undef WIDTH_
#undef CHANNELS_
}