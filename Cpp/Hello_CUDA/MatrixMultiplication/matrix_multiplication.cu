#include "hip/hip_runtime.h"
#include "matrix_multiplication.cuh"


#define TILE_WIDTH 32


void matrix_multiplication( float* m, float* n, float* out_mat, int width) {
    
    int size = width * width * sizeof(float);
    float* md, *nd, *od;
    
    // Transfrer m and n to device memory
    hipMalloc( (void**) &md, size);
    hipMemcpy( md, m, size, hipMemcpyHostToDevice);
    hipMalloc( (void**) &nd, size);
    hipMemcpy( nd, n, size, hipMemcpyHostToDevice);

    // allocate output matrix on device
    hipMalloc( (void**) &od, size);

    // kernel invocation
    dim3 dimBlock( TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid( width/TILE_WIDTH,width/TILE_WIDTH);
    matrix_multiplication_kernel<<<dimGrid, dimBlock>>>( md, nd, od, width);


    // transfer output matrix from device to host
    hipMemcpy( out_mat, od, size, hipMemcpyDeviceToHost);

    // free device memory
    hipFree(md);
    hipFree(nd);
    hipFree(od);
}



__global__ void matrix_multiplication_kernel( float* md, float* nd, float* od, int width) {

    __shared__ float mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float nds[TILE_WIDTH][TILE_WIDTH];

    // cache block & thread indices for faster access
    int bx( blockIdx.x);    int tx( threadIdx.x);
    int by( blockIdx.y);    int ty( threadIdx.y);
    
    // identify row & column of the matrix element to work on
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    
    // stores the od element that is computed by the thread
    float v(0);

    // loop over md and nd tiles required to compute v
    for( int i=0; i<width/TILE_WIDTH; ++i) {
   
        // collaborative loading of md and nd tiles into shared memory
        mds[ty][tx] = md[row*width + (i*TILE_WIDTH + tx)];
        nds[ty][tx] = nd[ (i*TILE_WIDTH + ty)*width + col];
        __syncthreads();

        for( int j=0; j<TILE_WIDTH; ++j)
            v += mds[ty][j] * nds[j][tx];
        __syncthreads();
    }
    od[row*width + col] = v;
}


__global__ void matrix_multiplication_kernel_old( float* md, float* nd, float* od, int width) {

    // cache block & thread indices for faster access
    int bx( blockIdx.x);    int tx( threadIdx.x);
    int by( blockIdx.y);    int ty( threadIdx.y);
    
    
    // stores the od element that is computed by the thread
    float v(0);

    for( int i=0; i<width; ++i) {
        v +=  md[ty*width+i] * nd[i*width + tx];
    };
    od[ty*width+tx] = v;
}