#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include <hipcub/hipcub.hpp>

using namespace std;

//
// Block-sorting CUDA kernel
//
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void block_sort_kernel(int *d_in, int *d_out)
{
    using namespace hipcub;

    // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
    typedef hipcub::BlockLoad<
        int*, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_TRANSPOSE> BlockLoadT;
    typedef hipcub::BlockStore<
        int*, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_STORE_TRANSPOSE> BlockStoreT;
    typedef hipcub::BlockRadixSort<
        int, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    // Allocate type-safe, repurposable shared memory for collectives
    __shared__ union {
        typename BlockLoadT::TempStorage       load; 
        typename BlockStoreT::TempStorage      store; 
        typename BlockRadixSortT::TempStorage  sort;
    } temp_storage; 

    // Obtain this block's segment of consecutive keys (blocked across threads)
    int thread_keys[ITEMS_PER_THREAD];
    int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);   
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, thread_keys);
    
    __syncthreads();    // Barrier for smem reuse
    // Collectively sort the keys
    BlockRadixSortT(temp_storage.sort).Sort(thread_keys);
    __syncthreads();    // Barrier for smem reuse
    // Store the sorted segment 
    BlockStoreT(temp_storage.store).Store(d_out + block_offset, thread_keys);
}



// Helper function for using CUDA to add vectors in parallel.
void block_sort(int *o, const int *m, int m_sz)
{
    int* d_o, *d_m;
    int m_bytes= m_sz*sizeof(int);


    hipMalloc((void**)&d_o, m_bytes);
    hipMalloc((void**)&d_m, m_bytes);
    hipMemcpy( d_m, m, m_bytes, hipMemcpyHostToDevice);

    
#define N_THREADS_P_BLOCK 128
#define N_KEYS_P_THREAD 16

    const dim3 dimBlock( N_THREADS_P_BLOCK);
    const dim3 dimGrid( (m_sz/N_KEYS_P_THREAD-1)/dimBlock.x+1);

    cout << "dimBlock.x = " << dimBlock.x << "\n"
            "dimGrid.x  = " << dimGrid.x << endl;
 
    // launch a block-sorting kernel in which each block of N_THREADS threads 
    // sorts segments of N_KEYS_P_THREAD*N_THREADS keys
    block_sort_kernel<N_THREADS_P_BLOCK, N_KEYS_P_THREAD><<<dimGrid, dimBlock>>>(d_m, d_o); 

    hipMemcpy( o, d_o, m_bytes, hipMemcpyDeviceToHost);

    hipFree(d_o);
    hipFree(d_m);

#undef N_THREADS
}

