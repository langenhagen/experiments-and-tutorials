#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <conio.h>




inline int block_num( int thread_num, int block_size) {
	return (thread_num-1)/block_size+1;
}



/// todo doc
__device__
float3 rgb2xyz_px( uchar3 rgb) {
    float3 ret;
    
    rgb.x = rgb.x / 255.0f;
    rgb.y = rgb.y / 255.0f;
    rgb.z = rgb.z / 255.0f;

    ret.x = 0.412453f * rgb.x + 0.357580f * rgb.y + 0.180423f * rgb.z;
    ret.y = 0.212671f * rgb.x + 0.715160f * rgb.y + 0.072169f * rgb.z;
    ret.z = 0.019334f * rgb.x + 0.119193f * rgb.y + 0.950227f * rgb.z;

    return ret;
}

/// TODO doc
__device__
float3 xyz2lab_px( float3 xyz) {
    float3 ret;
    
    xyz.x /= 0.950456f;
	float y3 = __expf( __logf(xyz.y)/3.0f);
	xyz.z /= 1.088754f;

	

	xyz.x = xyz.x>0.008856f ? __expf (__logf (xyz.x)/3.0f) : (7.787f * xyz.x + 0.13793f);
	xyz.y = xyz.y>0.008856f ? y3 : 7.787f * xyz.y + 0.13793f;
	xyz.z = xyz.z>0.008856f ? xyz.z /= __expf(__logf(xyz.z)/3.0f) : (7.787f * xyz.z + 0.13793f);
	
	ret.x = xyz.y > 0.008856f ? (116.0f *y3 - 16.0f) : 903.3f * xyz.y; // l
	ret.y = (xyz.x - xyz.y) * 500.0f; // a
	ret.z = (xyz.y - xyz.z) * 200.0f; // b
	
    return ret;
}


__device__
float3 rgb2lab_px( uchar3 rgb) {

    return xyz2lab_px( rgb2xyz_px(rgb));
}


__global__ 
void rgb2cielab(uchar3* in_image, float3* out_image, int width, int height) {
	int block_offset = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x;
	int offset = block_offset + threadIdx.x + threadIdx.y * width;

	uchar3 pixel = in_image[offset];


	out_image[offset]= rgb2lab_px( pixel);

    //manage overlapping
}



// Helper function for using CUDA to add vectors in parallel.
hipError_t rgb2cielab()
{
    hipError_t ret;

    // Choose which GPU to run on, change this on a multi-GPU system.
    ret = hipSetDevice(0);
    if (ret != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        // call hipFree() ...
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    ret = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");

    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");

    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");

    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");

    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");

    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));

    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");

    }


    return cudaStatus;
}




int main()
{
    cout << "

	getch();
    return 0;
}