#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <assert.h>
#include <conio.h>
#include <limits>
#include <random>
#include <time.h>

#include <boost/chrono.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>

using namespace std;
using namespace boost::chrono;


#define NSIZE 1024


template< int SIZE>
__global__
void normalize_kernel( float* o, float* i, int size) {

    //// collaboratively load the array
    //__shared__ float i_s[SIZE];
    //i_s[threadIdx.x] = i[ threadIdx.x];
    //__syncthreads();


    //float mn = i_s[0];
    //float mx = i_s[0];
    //
    //// find min / max
    //// could be made more divide&conquerish
    //for( int a=1; a<size; ++a) {
    //    mn = min( i_s[a], mn);
    //    mx = max( i_s[a], mx);
    //}

    //float scale = mx - mn;
    //o[threadIdx.x] = i_s[threadIdx.x]/scale + mn;
    o[threadIdx.x] = threadIdx.x;
}


void normalize_h( float* o, float* i, int size) {

    float min = i[0];
    float max = i[0];
    
    // find min / max
    for( int a=1; a<size; ++a) {
        if ( i[a] < min)
            min = i[a];
        else if( i[a] > max)
            max = i[a];
    }
    

    // normalize
    float scale = max - min;
    for( int a=1; a<size; ++a) {
        o[a] = i[a]/scale + min;
    }
}



void normalize_d( float* o, float* i, int size) {

    float *d_o, *d_i;

    hipMalloc( (void**)&d_i, size*sizeof(float));
    hipMemcpy( d_i, i, size*sizeof(float), hipMemcpyHostToDevice);
    //hipMalloc( (void**)&d_o, size*sizeof(float));
    //
    //dim3 dimBlock(NSIZE,1,1);
    //dim3 dimGrid(1,1,1);
    //
    //normalize_kernel<NSIZE><<<dimGrid, dimBlock>>>( o, i, size);

//    hipMemcpy( o, d_o, size*sizeof(float), hipMemcpyDeviceToHost);

    //hipFree( d_o);
    hipFree( d_i);

}



struct normalize_functor{
    
    const float scaler;
    const float addend;

    normalize_functor(float _scaler, float _addend) : scaler(_scaler), addend(_addend) 
    {} 


    __host__ __device__ 
    float operator()( const float& x) const { 
        return scaler * x + addend; 
    } 
};

void normalize_thrust( float* o, float* i, int size) {

    thrust::device_vector<float> i_d(i, i+size);
    

    //typedef thrust::pair<thrust::device_vector<float>::iterator, thrust::device_vector<float>::iterator> result_type;
    auto mnmx_d = thrust::minmax_element( i, i+size);
    
    float mn = *mnmx_d.first;
    float scale = *mnmx_d.second - mn;


    thrust::transform( i_d.begin(), i_d.end(), i_d.begin(), normalize_functor( scale, mn));
    
    thrust::copy(i_d.begin(), i_d.end(), o);
}


void main() {

    int n_runs = 100;
    steady_clock::time_point timer_start;
    unsigned int dur_h=0;
    unsigned int dur_d=0;
    unsigned int dur_t=0;

    srand( time(NULL));
    const int bin_size = NSIZE;

    cout << "Running " << n_runs << " times each.\n\n";


    float* arr  = (float*) malloc(bin_size*sizeof(float));
    float* narr = (float*) malloc(bin_size*sizeof(float));;

    for( int i=0; i<bin_size; ++i)
        arr[i] = rand();

    
    for( int i=0; i<n_runs; ++i) {       

        timer_start = steady_clock::now();
        normalize_h(narr, arr, bin_size);
        dur_h += round<microseconds>( steady_clock::now() - timer_start).count();

    }

    for( int i=0; i<n_runs; ++i) {

        timer_start = steady_clock::now();
        normalize_d( narr, arr, bin_size);
        dur_d += round<microseconds>( steady_clock::now() - timer_start).count();
    }


    for( int i=0; i<n_runs; ++i) {

        timer_start = steady_clock::now();
        normalize_thrust( narr, arr, bin_size);
        dur_t += round<microseconds>( steady_clock::now() - timer_start).count();
    }

    cout << "dur_h: " << dur_h << endl;
    cout << "dur_d: " << dur_d << endl;
    cout << "dur_t: " << dur_t << endl;

    hipDeviceReset();
    getch();
}

