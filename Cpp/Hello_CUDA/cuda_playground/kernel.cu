#include "hip/hip_runtime.h"
#define HELLO
/******************************************************************************
/* @file Playground kernel function. Have fun!
/*
/*
/* @author langenhagen
/* @version YYMMDD
/******************************************************************************/
#pragma once

///////////////////////////////////////////////////////////////////////////////
// INCLUDES project headers


///////////////////////////////////////////////////////////////////////////////
//INCLUDES C/C++ standard library (and other external libraries)

#include <cstdlib>
#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>
#include ""

#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/sort.h"
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>


//#include <barn_common.hpp> // << provokes linker errors!
#include <matrix.cuh>
#include <barn_cuda_common.cuh>

///////////////////////////////////////////////////////////////////////////////
// DEFINES and MACROS


///////////////////////////////////////////////////////////////////////////////
// NAMESPACE, CONSTANTS and TYPE DECLARATIONS/IMPLEMENTATIONS


using namespace std;
using namespace thrust;


typedef unsigned char uchar;



__global__ 
void play_kernel( Matrix<int>* m,
                  const int Kx,
                  const int Ky) {

    const int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if( tx+1 > m->n_cells())
        return;

    const int r = tx / m->cols;
    const int c = tx - r*m->cols;
    
    const int k_r = r * Ky / m->rows;
    const int k_c = c * Kx / m->cols;

    m->at(r,c) = k_r * Kx + k_c;
}


struct row_calculator : public thrust::unary_function<unsigned int /*input*/, unsigned int /*output*/> {

    unsigned int _n_cols;

    row_calculator( unsigned int n_cols) : _n_cols(n_cols)
    {}

    __host__ __device__
    unsigned int operator()(const unsigned int &i) const { 
        return i/_n_cols;
    }
};

struct col_calculator : public thrust::unary_function<unsigned int, unsigned int> {

    unsigned int _n_cols;

    col_calculator( unsigned int n_cols) : _n_cols(n_cols)
    {}

    __host__ __device__
    unsigned int operator()(const unsigned int &i) const { 
        unsigned int r = i/_n_cols;
        return i - r*_n_cols;
    }
};


void play() {
    Matrix<int> segmat( 25, 10);
    Matrix<int> *d_segmat = segmat.h2d();

    // create segment mat
    play_kernel<<< (segmat.rows*segmat.cols-1)/1024+1, 1024>>>( d_segmat, 5, 2);
    segmat.d2h( d_segmat, false);

    device_vector<int> segvec_d( segmat.data, segmat.data+segmat.n_cells()); // aka keys

    for( auto it=segvec_d.begin(); it!=segvec_d.end(); ++it)
        cout << *it << "\t";


    // constant iterator for count of segment size sp�ter
    constant_iterator<int> const_beg_it(1);
    constant_iterator<int> const_beg_end( const_beg_it + segmat.n_cells());
    
    cout << "const iter: \n"
            "const_beg_it[0]: " << const_beg_it[0] << "\n"
            "const_beg_it[1]: " << const_beg_it[1] << "\n"
            "const_beg_it[2]: " << const_beg_it[2] << "\n";
            

    // hiilfs-gn�rdel f�r die row/col iterators: countet alle cells durch _und_ist_�nderbar_
    device_vector<unsigned int> value_mappings( segmat.rows*segmat.cols);
    sequence(value_mappings.begin(), value_mappings.end());
 
    /*
    // hiilfs-iterator f�r row/col iterators: countet alle data cells durch
    counting_iterator<int> cnt_beg_it(0);
    counting_iterator<int> cnt_end_it( cnt_beg_it + segmat.n_cells());

    cout << "counting iter: \n"
            "cnt_beg_it[0]: " << cnt_beg_it[0] << "\n"
            "cnt_beg_it[1]: " << cnt_beg_it[1] << "\n"
            "cnt_beg_it[2]: " << cnt_beg_it[2] << "\n";*/

    
    
    // retrieves row / column for given data element

  //  typedef transform_iterator< row_calculator, thrust::counting_iterator<int>> row_it_t;
  //  typedef transform_iterator< col_calculator, thrust::counting_iterator<int>> col_it_t;

    auto row_beg_it = make_transform_iterator( counting_iterator<int>(0), row_calculator( segmat.cols));
    auto row_end_it = make_transform_iterator( value_mappings.end(),   row_calculator( segmat.cols));
    auto col_beg_it = make_transform_iterator( counting_iterator<int>(0), col_calculator( segmat.cols));
    auto col_end_it = make_transform_iterator( value_mappings.end(),   col_calculator( segmat.cols));


    cout << "row iter (with " << segmat.cols << " cols): \n"
            "row_beg_it[ 0]: " << row_beg_it[ 0] << "\n"
            "row_beg_it[ 1]: " << row_beg_it[ 1] << "\n"
            "row_beg_it[ 2]: " << row_beg_it[ 2] << "\n"
            "row_beg_it[12]: " << row_beg_it[12] << "\n"
            "row_beg_it[17]: " << row_beg_it[17] << "\n"
            "row_beg_it[29]: " << row_beg_it[29] << "\n";

    cout << "col iter (with " << segmat.cols << " cols): \n"
            "col_beg_it[ 0]: " << col_beg_it[ 0] << "\n"
            "col_beg_it[ 1]: " << col_beg_it[ 1] << "\n"
            "col_beg_it[ 2]: " << col_beg_it[ 2] << "\n"
            "col_beg_it[12]: " << col_beg_it[12] << "\n"
            "col_beg_it[17]: " << col_beg_it[17] << "\n"
            "col_beg_it[29]: " << col_beg_it[29] << "\n";


    /*
    auto values_beg_it = make_zip_iterator( make_tuple( row_beg_it, col_beg_it));
    auto values_end_it = make_zip_iterator( make_tuple( row_end_it, col_end_it));
    */

    

    sort_by_key(segvec_d.begin(), segvec_d.end(), value_mappings.begin());

    for( int i=0; i<segmat.rows*segmat.cols; ++i) {
        
        cout << "i: " << i << "  segment: " << segvec_d[i] << "  row: " << row_beg_it[value_mappings[i]] << "  col: " << col_beg_it[value_mappings[i]] << "\n";
        
    }



}




__global__ 
void play_kernel2() {
    
}

void play2() {

}



///////////////////////////////////////////////////////////////////////////////
// WARMUP KERNEL AND INVOCATION FUNCTION

__global__ 
void warmup_kernel() {
    printf( "Warming up... ");
}

void warmup() {
    warmup_kernel<<<1,1>>>();
    hipDeviceSynchronize();
    printf("Done.\n");
}

///////////////////////////////////////////////////////////////////////////////

//struct myfunctor {
//    
//    unsigned char x;
//
//    myfunctor( unsigned char _x) : x(_x)
//    {}
//
//    __host__ __device__
//    unsigned char operator()(const unsigned char &c) const { 
//        return x-c;
//    }
//};