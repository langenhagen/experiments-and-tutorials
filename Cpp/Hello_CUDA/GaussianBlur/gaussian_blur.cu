#include "hip/hip_runtime.h"
/******************************************************************************
/* @file Impl of gaussian_blur.cuh
/* 
/* There can be a lot optmized (e.g. for separable filters) but the current
/* implementation is quite straight forward and simple,  leading to
/* okay-ish results.
/* 
/* TODO rename 
/*
/* @author langenhagen
/* @version 141212
/******************************************************************************/

///////////////////////////////////////////////////////////////////////////////
// INCLUDES project headers

#include "gaussian_blur.cuh"

///////////////////////////////////////////////////////////////////////////////
//INCLUDES C/C++ standard library (and other external libraries)

#include <math.h>  // expf
#include <cstdlib> // malloc
#include <iostream>

///////////////////////////////////////////////////////////////////////////////
// KERNEL FUNCTIONS 

/**
 */
template<int RADIUS, int CHANNELS> __global__ 
void convolve_kernel( float* m,
                      float* o,
                      int rows,
                      int cols,
                      float* convolution_kernel) {

#if defined(KERNEL_WIDTH)
#error Macro KERNEL_WIDTH is already defined.
#endif

#define KERNEL_WIDTH (2*RADIUS+1)
        
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;
    
    // collaboratively create the kernel
    __shared__ float kernel_s[KERNEL_WIDTH][KERNEL_WIDTH];
    kernel_s[ty][tx] = convolution_kernel[ ty*KERNEL_WIDTH + tx];
    
    __syncthreads();

    cols *= CHANNELS;
    const int r = (blockIdx.y*KERNEL_WIDTH + ty);
	const int c = (blockIdx.x*KERNEL_WIDTH + tx)*CHANNELS + tz;
    
    if( r >= rows || c >= cols)
		return;

    const int height = rows-1;
    const int width  = cols-CHANNELS;
    float v(0.0f);

#pragma unroll
    for (int i=-RADIUS; i<=RADIUS; ++i)   // rows
#pragma unroll
    for (int j=-RADIUS; j<=RADIUS; ++j) { // cols
			
        // clamp filter to image borders
		const int m_r = min(max(r+i, 0), height);
		const int m_c = min(max(c+j*CHANNELS, tz), width+tz);
        
		v += m[m_r*cols+m_c] * kernel_s[i+RADIUS][j+RADIUS];
	}
    
    o[r*cols+c] = v;

#undef KERNEL_WIDTH
}


///////////////////////////////////////////////////////////////////////////////
// HELPER FUNCTIONS 

/*
 */
__host__ 
float* prepare_gaussian_kernel( int kernel_radius, float sigma) {
    
    const int width(kernel_radius+kernel_radius+1);
    float* ret = (float*)malloc( width*width*sizeof(float));
    float sum(0);
    
    // calculate weights
    float reciprocal_two_sigma_squared = 1/(2.0f*sigma*sigma);
    for( int r=-kernel_radius; r<=kernel_radius; ++r)
    for( int c=-kernel_radius; c<=kernel_radius; ++c) {
    
        float weight = expf( -(c*c+r*r) * reciprocal_two_sigma_squared);
        int idx = (r+kernel_radius)*width + c+kernel_radius;
        ret[idx] = weight;
        sum += weight;
    }
    
    // normalize
    sum = 1.0f/sum;
    for( int i=0; i<width*width; ++i)
        ret[i] *= sum;
    
    return ret;
}


///////////////////////////////////////////////////////////////////////////////
// TEST FUNCTIONS

/**
 */
__host__
void test_convolution_17x17x3_kernel( int rows, 
                                      int cols,
                                      const float* m,
                                      float* o,
                                      float sigma) {

#define RADIUS_ 8
#define WIDTH_ (RADIUS_ + RADIUS_ + 1)
#define CHANNELS_ 3

    float* d_m, *d_o, *kernel, *d_kernel;
    int m_size = rows*cols*CHANNELS_*sizeof(float);
    int k_size = WIDTH_*WIDTH_*sizeof(float);

    kernel = prepare_gaussian_kernel( RADIUS_, sigma);


    // incorp texture memory //

    hipMalloc( (void**) &d_m, m_size);
    hipMemcpy( d_m, m, m_size, hipMemcpyHostToDevice);
    hipMalloc( (void**) &d_kernel, k_size);
    hipMemcpy( d_kernel, kernel, k_size, hipMemcpyHostToDevice);
    hipMalloc( (void**)&d_o, m_size);


    const dim3 dimBlock( WIDTH_, WIDTH_, CHANNELS_);
    const dim3 dimGrid( (cols-1)/WIDTH_+1, (rows-1)/WIDTH_+1);

    convolve_kernel<RADIUS_,CHANNELS_><<<dimGrid, dimBlock>>>(
        d_m,
        d_o,
        rows, 
        cols,
        d_kernel);


    hipMemcpy( o, d_o, m_size, hipMemcpyDeviceToHost);

    hipFree(d_kernel);
    hipFree(d_m);
    hipFree(d_o);

#undef RADIUS_
#undef WIDTH_
#undef CHANNELS_ 
}